#include <hip/hip_runtime.h>

#include <iostream>

typedef float (* fp)(float, float, float4);

struct functor
{
    float c0, c1;
    fp f;

    __device__ __host__
    functor(float _c0, float _c1, fp _f) : c0(_c0), c1(_c1), f(_f) {};

    __device__ __host__
    float operator()(float4 x) { return f(c0, c1, x); };
};

__global__
void kernel(float c0, float c1, fp f, const float4 * x, float * y, int N)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    struct functor op(c0, c1, f);
    for(int i = tid; i < N; i  = blockDim.x * gridDim.x) {
        y[i] = op(x[i]);
    }
}

__device__ __host__
float f1 (float a, float b, float4 c)
{
    return a  + (b * c.x) + (b * c.y) + (b * c.z) + (b * c.w);
}

__device__ __host__
float f2 (float a, float b, float4 c)
{
    return a + b + c.x + c.y + c.z + c.w;
}

__constant__ fp function_table[] = {f1, f2};

int main(void)
{
    hipSetDevice(1);

    const float c1 = 1.0f, c2 = 2.0f;
    const int n = 20;
    float4 vin[n];
    float vout1[n], vout2[n];
    for(int i=0, j=0; i<n; i++ ) {
        vin[i].x = j  ; vin[i].y = j  ;
        vin[i].z = j  ; vin[i].w = j  ;
    }

    float4 * _vin;
    float * _vout1, * _vout2;
    size_t sz4 = sizeof(float4) * size_t(n);
    size_t sz1 = sizeof(float) * size_t(n);
    hipMalloc((void **)&_vin, sz4);
    hipMalloc((void **)&_vout1, sz1);
    hipMalloc((void **)&_vout2, sz1);
    hipMemcpy(_vin, &vin[0], sz4, hipMemcpyHostToDevice);

    fp funcs[2];
    hipMemcpyFromSymbol(&funcs, HIP_SYMBOL("function_table"), 2 * sizeof(fp));

    kernel<<<1,32>>>(c1, c2, funcs[0], _vin, _vout1, n);
    hipMemcpy(&vout1[0], _vout1, sz1, hipMemcpyDeviceToHost);

    kernel<<<1,32>>>(c1, c2, funcs[1], _vin, _vout2, n);
    hipMemcpy(&vout2[0], _vout2, sz1, hipMemcpyDeviceToHost);

    struct functor func1(c1, c2, f1), func2(c1, c2, f2);
    for(int i=0; i<n; i++) {
        printf("- %6.f %6.f (%6.f,%6.f,%6.f,%6.f ) %6.f %6.f %6.f %6.f\n",
                i, c1, c2, vin[i].x, vin[i].y, vin[i].z, vin[i].w,
                vout1[i], func1(vin[i]), vout2[i], func2(vin[i]));
    }

    return 0;
}